/*
nvcc -arch=sm_35 -o delay delay.cu
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_DELAY 30

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

#define APPRX_CLKS_PER_SEC 1000000000ULL
__global__ void delay_kernel(unsigned seconds){

  unsigned long long dt = clock64();
  while (clock64() < (dt + (seconds*APPRX_CLKS_PER_SEC)));
}

int main(int argc, char *argv[]){

  unsigned delay_t = 2; // seconds, approximately
  unsigned delay_t_r;
  if (argc > 1) delay_t_r = atoi(argv[1]);
  if ((delay_t_r > 0) && (delay_t_r < MAX_DELAY)) delay_t = delay_t_r;
  unsigned long long difft = dtime_usec(0);
  delay_kernel<<<1,1>>>(delay_t);
  hipDeviceSynchronize();
  //cudaCheckErrors("kernel fail");
  difft = dtime_usec(difft);
  printf("kernel duration: %fs\n", difft/(float)USECPSEC);
  return 0;
}
