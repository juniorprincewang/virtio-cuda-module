#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *a)
{
	int tx = threadIdx.x;
	a[tx] = a[tx] + 8;
}


int main()
{
	int *d_a;
	int *a;
	dim3 threads(10,1);
	dim3 blocks(1,1);
	int nstreams = 5;
	int i;
	int nreps = 10;
	// create CUDA event handles
    // use blocking sync
    hipEvent_t start_event, stop_event;
	float elapsed_time;   // timing variables

    printf("\nStarting Test\n");

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));

    for (i = 0; i < nstreams; i++)
    {
        hipStreamCreate(&(streams[i]));
    }


	a=(int*)malloc(sizeof(int)*2);
	a[0]=1;
	a[1]=2;
	hipMalloc((void**)&d_a, sizeof(int)*2);
	hipMemcpyAsync(d_a, a, sizeof(int)*2, hipMemcpyHostToDevice, streams[0]);
	printf("a[0] = %d, a[1] = %d\n", a[0], a[1]);

	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
	// start
	hipEventRecord(start_event, 0);
	for (i = 0; i < nreps; i++) {
		kernel<<<blocks, threads, 0, streams[0]>>>(d_a);
	}
	// end
	hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
    printf("non-streamed:\t%.2f\n", elapsed_time / nreps);

	hipMemcpy(a, d_a, sizeof(int)*2, hipMemcpyDeviceToHost);
	printf("a[0] = %d, a[1] = %d\n", a[0], a[1]);

    // release resources
    for (i = 0; i < nstreams; i++)
    {
        hipStreamDestroy(streams[i]);
    }

	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
	hipFree(d_a);
	return 0;
}