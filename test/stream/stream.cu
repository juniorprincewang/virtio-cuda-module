#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
}

int checkResult(float *data, const int n, const float x)
{
    for (int i = 0; i < n; i++)
    {
        if (data[i] != x)
        {
            printf("Error! data[%d] = %f, ref = %f\n", i, data[i], x);
            return 0;
        }
    }

    return 1;
}


int main()
{
	float *d_a;
	float *h_a;
	int nstreams = 2;
	int i,j;
	int nreps = 10;
    int num = 1 << 10;
    int nbytes = num * sizeof(float);
    float value = 16;
	// create CUDA event handles
    // use blocking sync
    hipEvent_t start_event, stop_event;
	float elapsed_time;   // timing variables

    printf("\nStarting Test\n");

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));

    for (i = 0; i < nstreams; i++)
    {
        hipStreamCreate(&(streams[i]));
    }

    h_a=(float*)malloc(nbytes);
    memset(h_a, 0, nbytes);

	dim3 block = dim3(32,1,1);
    dim3 grid  = dim3((num + block.x - 1) / block.x);

	hipMalloc((void**)&d_a, nbytes);

	// cudaMemcpyAsync(d_a, a, sizeof(float)*2, cudaMemcpyHostToDevice, streams[0]);
    hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
	// start
	hipEventRecord(start_event, 0);
	for (i = 0; i < nreps; i++) {
        for (j=0; j<nstreams; j++) {
		  kernel<<<grid, block, 0, streams[j]>>>(d_a, value);
        }
	}
	// end
	hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
    printf("non-streamed:\t%.2f\n", elapsed_time / nreps);

	hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);

    int bFinalResults = checkResult(h_a, num, nstreams * nreps * value);
    printf("result:%s\n", bFinalResults? "PASS" : "FAILED");
    // release resources
    for (i = 0; i < nstreams; i++)
    {
        hipStreamDestroy(streams[i]);
    }

	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
	hipFree(d_a);
    free(h_a);
	return 0;
}
