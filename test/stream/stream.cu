#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
    printf("%f+g_data[%d]=%f\n", value, idx, g_data[idx]);
}

int checkResult(float *data, const int n, const float x)
{
    for (int i = 0; i < n; i++)
    {
        if (data[i] != x)
        {
            printf("Error! data[%d] = %f, ref = %f\n", i, data[i], x);
            return 0;
        }
    }

    return 1;
}


int main()
{
	int *d_a;
	int *a;
	dim3 threads(10,1);
	dim3 blocks(1,1);
	int nstreams = 5;
	int i;
	int nreps = 10000;
	// create CUDA event handles
    // use blocking sync
    hipEvent_t start_event, stop_event;
	float elapsed_time;   // timing variables

    printf("\nStarting Test\n");

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));

    for (i = 0; i < nstreams; i++)
    {
        hipStreamCreate(&(streams[i]));
    }


	dim3 block = dim3(4);
    dim3 grid  = dim3((num + block.x - 1) / block.x);

	hipMalloc((void**)&d_a, sizeof(int)*2);
	hipMemcpyAsync(d_a, a, sizeof(int)*2, hipMemcpyHostToDevice, streams[0]);
	printf("a[0] = %d, a[1] = %d\n", a[0], a[1]);

	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
	// start
	hipEventRecord(start_event, 0);
	for (i = 0; i < nreps; i++) {
		kernel<<<grid, block, 0, streams[0]>>>(d_a);
	}
	// end
	hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
    printf("non-streamed:\t%.2f\n", elapsed_time / nreps);

	hipMemcpy(a, d_a, sizeof(int)*2, hipMemcpyDeviceToHost);
	printf("a[0] = %d, a[1] = %d\n", a[0], a[1]);

    // release resources
    for (i = 0; i < nstreams; i++)
    {
        hipStreamDestroy(streams[i]);
    }

	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
	hipFree(d_a);
	return 0;
}
