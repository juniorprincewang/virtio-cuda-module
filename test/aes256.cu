#include "hip/hip_runtime.h"
//
//  CasAES_CUDA.c
//  CasAES_CUDA
//  Created by Carter McCardwell on 11/11/14.
//

#include <stdint.h>
#include <stdio.h>
#include <time.h>
#include <string.h>

#include <hip/hip_runtime.h>

const int Nb_h = 4;
const int Nr_h = 14;
const int Nk_h = 8;

const uint8_t s_h[256] = {
		0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
		0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
		0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
		0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
		0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
		0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
		0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
		0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
		0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
		0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
		0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
		0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
		0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
		0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
		0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
		0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

uint8_t Rcon_h[256] = {
		0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a,
		0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39,
		0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a,
		0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8,
		0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef,
		0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc,
		0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b,
		0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3,
		0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94,
		0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20,
		0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35,
		0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f,
		0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04,
		0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63,
		0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd,
		0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d
};

__constant__ uint8_t s[256];
__constant__ int Nb;
__constant__ int Nr;
__constant__ int Nk;
__constant__ uint32_t ek[60];

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void cudaDevAssist(hipError_t code, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"cudaDevAssistant: %s %d\n", hipGetErrorString(code), line);
		if (abort) exit(code);
	}
}

uint32_t sw(uint32_t word)
{
	union {
		uint32_t word;
		uint8_t bytes[4];
	} subWord  __attribute__ ((aligned));
	subWord.word = word;

	subWord.bytes[3] = s_h[subWord.bytes[3]];
	subWord.bytes[2] = s_h[subWord.bytes[2]];
	subWord.bytes[1] = s_h[subWord.bytes[1]];
	subWord.bytes[0] = s_h[subWord.bytes[0]];

	return subWord.word;
}

__device__ void sb(uint8_t* in)
{
	for (int i = 0; i < 32; i++) { in[i] = s[in[i]]; }
}

__device__ void sb_st(uint8_t* in)
{
	for (int i = 0; i < 16; i++) { in[i] = s[in[i]]; }

}

__device__ void mc(uint8_t* arr)
{
	for (int i = 0; i < 4; i++)
	{
		uint8_t a[4];
		uint8_t b[4];
		uint8_t c;
		uint8_t h;
		for(c=0;c<4;c++) {
			a[c] = arr[(4*c+i)];
			h = (uint8_t)((signed char)arr[(4*c+i)] >> 7);
			b[c] = arr[(4*c+i)] << 1;
			b[c] ^= 0x1B & h;
		}
		arr[(i)] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1];
		arr[(4+i)] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2];
		arr[(8+i)] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3];
		arr[(12+i)] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0];
	}

}

__device__ void sr(uint8_t* arr)
{
	uint8_t out[16];
	//On per-row basis (+1 shift ea row)
	//Row 1
	out[0] = arr[0];
	out[1] = arr[1];
	out[2] = arr[2];
	out[3] = arr[3];
	//Row 2
	out[4] = arr[5];
	out[5] = arr[6];
	out[6] = arr[7];
	out[7] = arr[4];
	//Row 3
	out[8] = arr[10];
	out[9] = arr[11];
	out[10] = arr[8];
	out[11] = arr[9];
	//Row 4
	out[12] = arr[15];
	out[13] = arr[12];
	out[14] = arr[13];
	out[15] = arr[14];

	for (int i = 0; i < 16; i++)
	{
		arr[i] = out[i];
	}
}

uint32_t rw(uint32_t word)
{
	union {
		uint8_t bytes[4];
		uint32_t word;
	} subWord  __attribute__ ((aligned));
	subWord.word = word;

	uint8_t B0 = subWord.bytes[3], B1 = subWord.bytes[2], B2 = subWord.bytes[1], B3 = subWord.bytes[0];
	subWord.bytes[3] = B1; //0
	subWord.bytes[2] = B2; //1
	subWord.bytes[1] = B3; //2
	subWord.bytes[0] = B0; //3

	return subWord.word;
}

void K_Exp(uint8_t* pk, uint32_t* out)
{
	int i = 0;
	union {
		uint8_t bytes[4];
		uint32_t word;
	} temp __attribute__ ((aligned));
	union {
		uint8_t bytes[4];
		uint32_t word;
	} univar[60] __attribute__ ((aligned));

	for (i = 0; i < Nk_h; i++)
	{
		univar[i].bytes[3] = pk[i*4];
		univar[i].bytes[2] = pk[i*4+1];
		univar[i].bytes[1] = pk[i*4+2];
		univar[i].bytes[0] = pk[i*4+3];
	}

	for (i = Nk_h; i < Nb_h*(Nr_h+1); i++)
	{
		temp.word = univar[i-1].word;
		if (i % Nk_h == 0)
		{
			temp.word = (sw(rw(temp.word)));
			temp.bytes[3] = temp.bytes[3] ^ (Rcon_h[i/Nk_h]);
		}
		else if (Nk_h > 6 && i % Nk_h == 4)
		{
			temp.word = sw(temp.word);
		}
		if (i-4 % Nk_h == 0)
		{
			temp.word = sw(temp.word);
		}
		univar[i].word = univar[i-Nk_h].word ^ temp.word;
	}
	for (i = 0; i < 60; i++)
	{
		out[i] = univar[i].word;
	}
}

__device__ void ark(uint8_t* state, int strD, uint32_t* eK)
{
	union {
		uint32_t word;
		uint8_t bytes[4];
	} kb[4] __attribute__ ((aligned));

	kb[0].word = eK[strD];
	kb[1].word = eK[strD+1];
	kb[2].word = eK[strD+2];
	kb[3].word = eK[strD+3];

	for (int i = 0; i < 4; i++)
	{
		state[i] = state[i] ^ kb[i].bytes[3];
		state[i+4] = state[i+4] ^ kb[i].bytes[2];
		state[i+8] = state[i+8] ^ kb[i].bytes[1];
		state[i+12] = state[i+12] ^ kb[i].bytes[0];
	}
}

__global__ void cudaRunner(uint8_t *in)
{
	uint8_t state[16];
  int localid = blockDim.x * blockIdx.x + threadIdx.x; //Data is shifted by 16 * ID of worker
  for (int i = 0; i < 16; i++) { state[i] = in[(localid*16)+i]; }

	ark(state, 0, ek);
	for (int i = 1; i < 14; i++)
	{
		sb_st(state);
		sr(state);
		mc(state);
		ark(state, i*Nb, ek);
	}

	sb_st(state);
	sr(state);
	ark(state, Nr*Nb, ek);

	for (int i = 0; i < 16; i++) { in[(localid*16)+i] = state[i]; }
}

int main(int argc, const char * argv[])
{
	printf("CasAES_CUDA Hyperthreaded AES-256 Encryption for CUDA processors - compiled 3/25/2015 Rev. 4\nCarter McCardwell, Northeastern University NUCAR - http://coe.neu.edu/~cmccardw - mccardwell.net\nPlease Wait...\n");

  clock_t c_start, c_stop;
  c_start = clock();

	FILE *infile;
	FILE *keyfile;
	FILE *outfile;

	infile = fopen(argv[2], "r");
    if (infile == NULL) { printf("error (infile)\n"); return(1); }
	keyfile = fopen(argv[3], "rb");
    if (keyfile == NULL) { printf("error (keyfile)\n"); return(1); }
	outfile = fopen(argv[4], "w");
    if (outfile == NULL) { printf("error (outfile permission error, run with sudo)\n"); return(1); }

    //Hex info, or ASCII
    bool hexMode = false;
    if (strcmp(argv[1], "h") == 0) { hexMode = true; }
    else if (strcmp(argv[1], "a") == 0) { hexMode = false; }
    else { printf("error: first argument must be \'a\' for ASCII interpretation or \'h\' for hex interpretation\n"); return(1); }

	uint8_t key[32];
	uint32_t ek_h[60];

	for (int i = 0; i < 32; i++)
	{
		fscanf(keyfile, "%x", &key[i]);
	}

	K_Exp(key, ek_h);

	//send constants to GPU
	hipSetDevice(0);
	cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nk), &Nk_h, sizeof(int), 0, hipMemcpyHostToDevice), 535, true);
	cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nr), &Nr_h, sizeof(int), 0, hipMemcpyHostToDevice), 543, true);
	cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nb), &Nb_h, sizeof(int), 0, hipMemcpyHostToDevice), 903, true);
	cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(s), &s_h, 256*sizeof(uint8_t), 0, hipMemcpyHostToDevice), 920, true);
	cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(ek), &ek_h, 60*sizeof(uint32_t), 0, hipMemcpyHostToDevice), 823, true);
	hipDeviceSynchronize();

	const int BLOCKS = -1; //Not used
	const int RUNNING_THREADS = 512;

	uint8_t *devState = NULL;
	cudaDevAssist(hipMalloc((void**)&devState, RUNNING_THREADS*16*sizeof(uint8_t)), 425, true);

	uint8_t states[RUNNING_THREADS][16] = { 0x00 };
  int ch = 0;
	int spawn = 0;
	int end = 1;
	while (end)
	{
		spawn = 0;
		for (int i = 0; i < RUNNING_THREADS; i++) //Dispatch many control threads that will report back to main (for now 5x) - 1 worker per state
		{
            spawn++;
			for (int ix = 0; ix < 16; ix++)
			{
                if (hexMode)
                {
                    if (fscanf(infile, "%x", &states[i][ix]) != EOF) { ; }
                    else
                    {
                        if (ix > 0) { for (int ixx = ix; ixx < 16; ixx++) { states[i][ixx] = 0x00; } }
                        else { spawn--; }
                        i = RUNNING_THREADS + 1;
                        end = 0;
                        break;
                    }
                }
                else
                {
                    ch = getc(infile);
                    if (ch != EOF) { states[i][ix] = ch; }
                    else
                    {
                        if (ix > 0) { for (int ixx = ix; ixx < 16; ixx++) { states[i][ixx] = 0x00; } }
                        else { spawn--; }
                        i = RUNNING_THREADS + 1;
                        end = 0;
                        break;
                    }
                }
			}
		}
		//arrange data correctly
		for (int i = 0; i < spawn; i++)
		{
			uint8_t temp[16];
			memcpy(&temp[0], &states[i][0], sizeof(uint8_t));
			memcpy(&temp[4], &states[i][1], sizeof(uint8_t));
			memcpy(&temp[8], &states[i][2], sizeof(uint8_t));
			memcpy(&temp[12], &states[i][3], sizeof(uint8_t));
			memcpy(&temp[1], &states[i][4], sizeof(uint8_t));
			memcpy(&temp[5], &states[i][5], sizeof(uint8_t));
			memcpy(&temp[9], &states[i][6], sizeof(uint8_t));
			memcpy(&temp[13], &states[i][7], sizeof(uint8_t));
			memcpy(&temp[2], &states[i][8], sizeof(uint8_t));
			memcpy(&temp[6], &states[i][9], sizeof(uint8_t));
			memcpy(&temp[10], &states[i][10], sizeof(uint8_t));
			memcpy(&temp[14], &states[i][11], sizeof(uint8_t));
			memcpy(&temp[3], &states[i][12], sizeof(uint8_t));
			memcpy(&temp[7], &states[i][13], sizeof(uint8_t));
			memcpy(&temp[11], &states[i][14], sizeof(uint8_t));
			memcpy(&temp[15], &states[i][15], sizeof(uint8_t));
			for (int c = 0; c < 16; c++) { memcpy(&states[i][c], &temp[c], sizeof(uint8_t)); }
		}

		//printf("\nCycle!: Spawn = %i", spawn);

		cudaDevAssist(hipMemcpy(devState, *states, spawn*16*sizeof(uint8_t), hipMemcpyHostToDevice), 426, true);
		cudaDevAssist(hipDeviceSynchronize(), 268, true);
		cudaRunner<<<1,spawn>>>(devState);

		cudaDevAssist(hipDeviceSynchronize(), 270, true);
		cudaDevAssist(hipMemcpy(*states, devState, spawn*16*sizeof(uint8_t), hipMemcpyDeviceToHost), 431, true);


		//Write results to out
		for (int i = 0; i < spawn; i++)
		{
			for (int ix = 0; ix < 4; ix++)
			{
				char hex[3];
				sprintf(hex, "%02x", states[i][ix]);
				for (int i = 0; i < 3; i++) { putc(hex[i], outfile); }
				sprintf(hex, "%02x", states[i][ix+4]);
				for (int i = 0; i < 3; i++) { putc(hex[i], outfile); }
				sprintf(hex, "%02x", states[i][ix+8]);
				for (int i = 0; i < 3; i++) { putc(hex[i], outfile); }
				sprintf(hex, "%02x", states[i][ix+12]);
				for (int i = 0; i < 3; i++) { putc(hex[i], outfile); }
			}
		}
	}
  c_stop = clock();
  float diff = (((float)c_stop - (float)c_start) / CLOCKS_PER_SEC ) * 1000;

  printf("Done - Time taken: %f ms\n", diff);
	hipFree(devState);
	hipDeviceReset();
	fclose(infile);
	fclose(outfile);
	fclose(keyfile);
	return 0;
}