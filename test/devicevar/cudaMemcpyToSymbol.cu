// test constant variable and cudaMemcpyToSymbol
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

__constant__ float dfactor;
__constant__ float dSecrets[8]={0.1, 0.1, 0.1,0.1,0.1,0.1,0.1,0.1};

__global__ void test(float *a, int size, int value)
{
    int idx = threadIdx.x;
    if(idx<size)
        a[idx] = dfactor;
}

__global__ void test2()
{
    float a = dfactor;
    a +=1.0;
}

int main(void)
{

    float factor=9.0f;
    float h_factor = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(dfactor), &factor, sizeof(float), 0, hipMemcpyHostToDevice);
    test2<<<1,1>>>();
    hipMemcpyFromSymbol(&h_factor, HIP_SYMBOL(dfactor), sizeof(float), 0, hipMemcpyDeviceToHost);
    printf("host factor = %f\n", h_factor);
    
    float *da;
    float ha=0;
    
    std::cout << "the original value is " << ha << std::endl;
    hipMalloc((void **)&da, sizeof(float));
    test<<<1,1>>>(da,1,2);
    hipMemcpy(&ha, da, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(da);
    hipDeviceSynchronize();
    std::cout << "the value is now " << ha << std::endl;
    
    return 0;
}
