#include <stdio.h>  
#include <hip/hip_runtime.h>  

#define checkCudaErrors(call) { \
  hipError_t err; \
  if ( (err = (call)) != hipSuccess) { \
    fprintf(stderr, "Got error %s at %s:%d\n", hipGetErrorString(err), \
        __FILE__, __LINE__); \
  } \
}

/* A very simple kernel function */
 __global__ void kernel(int *d_var) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_var[idx] += 10; 
} 
 
 int * host_p;  
 int * host_result;  
 int * dev_p;  
 
int main(void) {  
      // int ns = 4;
      // int ns = 1<<8;
        int ns = 1<<15;
      int data_size = ns * sizeof(int);
      
      /* Allocate host_p as pinned memory */
      checkCudaErrors( 
        hipHostAlloc((void**)&host_p, data_size, 
        hipHostMallocDefault) );  
      
      /* Allocate host_result as pinned memory */
      checkCudaErrors( 
        hipHostAlloc((void**)&host_result, data_size, 
        hipHostMallocDefault) );  
      /* Allocate dev_p on the device global memory */
      checkCudaErrors( 
        hipMalloc((void**)&dev_p, data_size) );  
      
      /* Initialise host_p*/
      for (int i=0; i<ns; i++){  
           host_p[i] = i + 1;  
      }  
      
      /* Transfer data to the device host_p .. dev_p */
      checkCudaErrors( 
        hipMemcpy(dev_p, host_p, data_size, hipMemcpyHostToDevice) );
      
    /* Now launch the kernel... */
    dim3 block, grid;
    // set kernel launch configuration
    block = dim3(32);
    grid  = dim3((ns + block.x - 1) / block.x);
    kernel<<<grid, block>>>(dev_p);  
    checkCudaErrors(hipGetLastError());
      
      /* Copy the result from the device back to the host */
      checkCudaErrors( 
        hipMemcpy(host_result, dev_p, data_size, hipMemcpyDeviceToHost) );
      
      printf("Check if no failures, then pass.\n");      
      /* and print the result */
      for (int i=0; i<ns; i++){  
            if (host_result[i] != i+11)
                printf("Failed result[%d] = %d\n", i, host_result[i]);  
      }
      /*
       * Now free the memory!
       */
      checkCudaErrors( hipFree(dev_p) );  
      checkCudaErrors( hipHostFree(host_p) );  
      checkCudaErrors( hipHostFree(host_result) );  
      
      return 0;  
 } 