#include <stdio.h>
#include <hip/hip_runtime.h>

/*
 * An example of using CUDA events to control asynchronous work launched on the
 * GPU. In this example, asynchronous copies and an asynchronous kernel are
 * used. A CUDA event is used to determine when that work has completed.
 */

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
}

int checkResult(float *data, const int n, const float x)
{
    for (int i = 0; i < n; i++)
    {
        if (data[i] != x)
        {
            printf("Error! data[%d] = %f, ref = %f\n", i, data[i], x);
            return 0;
        }
    }

    return 1;
}

int main(int argc, char *argv[])
{
    int devID = 0;
    hipDeviceProp_t deviceProps;
    CHECK(hipGetDeviceProperties(&deviceProps, devID));
    printf("> %s running on", argv[0]);
    printf(" CUDA device [%s]\n", deviceProps.name);

    int num = 1 << 24;
    int nbytes = num * sizeof(int);
    float value = 10.0f;

    // allocate host memory
    float *h_a = 0;
    CHECK(hipHostMalloc((void **)&h_a, nbytes, hipHostMallocDefault));
    memset(h_a, 0, nbytes);

    // allocate device memory
    float *d_a = 0;
    CHECK(hipMalloc((void **)&d_a, nbytes));
    CHECK(hipMemset(d_a, 255, nbytes));

    // set kernel launch configuration
    dim3 block = dim3(512);
    dim3 grid  = dim3((num + block.x - 1) / block.x);

    float ms; // elapsed time in milliseconds
    // create cuda event handles
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    // asynchronously issue work to the GPU (all to stream 0)
    CHECK(hipEventRecord(start));
    CHECK(hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    kernel<<<grid, block>>>(d_a, value);
    CHECK(hipMemcpyAsync(h_a, d_a, nbytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&ms, start, stop) );
    printf("Time for sequential transfer and execute (ms): %f\n", ms);
    
    /*
    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;

    while (cudaEventQuery(stop) == cudaErrorNotReady) {
        counter++;
    }

    // print the cpu and gpu times
    printf("CPU executed %lu iterations while waiting for GPU to finish\n",
           counter);
    */

    // check the output for correctness
    bool bFinalResults = (bool) checkResult(h_a, num, value);

    // release resources
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    CHECK(hipHostFree(h_a));
    CHECK(hipFree(d_a));

    CHECK(hipDeviceReset());
    printf("result:%d\n", bFinalResults);
    exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}
