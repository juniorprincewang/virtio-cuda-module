#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
    printf("%f+g_data[%d]=%f\n", value, idx, g_data[idx]);
}

int checkResult(float *data, const int n, const float x)
{
    for (int i = 0; i < n; i++)
    {
        if (data[i] != x)
        {
            printf("Error! data[%d] = %f, ref = %f\n", i, data[i], x);
            return 0;
        }
    }

    return 1;
}

int main()
{
    int devID=1;
    int count = 0;
    struct hipDeviceProp_t props;
    float *d_a=0;
    float *h_a=0;
    dim3 block, grid;
    int num = 1 << 4;
    int nbytes = num * sizeof(float);
    int value=16;
    int nStreams = 4;
    //test();

    hipGetDeviceCount(&count);
    printf("cuda count=%d\n", count); 
    // return 0;

    printf("[=] Before devID is %d\n",  devID);
    hipGetDevice(&devID);
    printf("[=] After devID is %d\n",  devID);
    printf("prop=%lu\n", sizeof(struct hipDeviceProp_t));  
    hipGetDeviceProperties(&props, devID);
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",devID, props.name, props.major, props.minor);
    // return 0;

    h_a=(float*)malloc(nbytes);
    memset(h_a, 0, nbytes);
    // start
    hipMalloc((void**)&d_a, nbytes);
    hipMemset(d_a, 0, nbytes);
    // set kernel launch configuration
    block = dim3(4);
    grid  = dim3((num + block.x - 1) / block.x);

    float ms; // elapsed time in milliseconds
    // create events and streams
    hipEvent_t startEvent, stopEvent, dummyEvent;
    hipStream_t stream[nStreams];
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventCreate(&dummyEvent);
    for (int i = 0; i < nStreams; ++i)
        hipStreamCreate(&stream[i]);

    hipEventRecord(startEvent,0);
    hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
    kernel<<<grid, block>>>(d_a, value);
    hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("Time for sequential transfer and execute (ms): %f\n", ms);

    bool bFinalResults = (bool) checkResult(h_a, num, value);
    printf("result:%d\n", bFinalResults);
    // end
    free(h_a);
    hipFree(d_a);
     // cleanup
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipEventDestroy(dummyEvent);
    for (int i = 0; i < nStreams; ++i)
        hipStreamDestroy(stream[i]);
    return 0;
}
