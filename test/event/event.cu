#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
    // printf("%f+g_data[%d]=%f\n", value, idx, g_data[idx]);
}

int checkResult(float *data, const int n, const float x)
{
    for (int i = 0; i < n; i++)
    {
        if (data[i] != x)
        {
            printf("Error! data[%d] = %f, ref = %f\n", i, data[i], x);
            return 0;
        }
    }

    return 1;
}

int main()
{
    int devID=1;
    int count = 0;
    struct hipDeviceProp_t props;
    float *d_a=0;
    float *h_a=0;
    int num = 1 << 4;
    int nbytes = num * sizeof(float);
    int value=16;
    int nStreams = 4;
    //test();

    hipGetDeviceCount(&count);
    printf("cuda count=%d\n", count); 
    // return 0;

    printf("[=] Before devID is %d\n",  devID);
    hipGetDevice(&devID);
    printf("[=] After devID is %d\n",  devID);
    printf("prop=%lu\n", sizeof(struct hipDeviceProp_t));  
    hipGetDeviceProperties(&props, devID);
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",devID, props.name, props.major, props.minor);
    // return 0;

    h_a=(float*)malloc(nbytes);
    memset(h_a, 0, nbytes);
    // start
    hipMalloc((void**)&d_a, nbytes);
    hipMemset(d_a, 0, nbytes);
    // set kernel launch configuration
    dim3 block = dim3(4);
    dim3 grid  = dim3((num + block.x - 1) / block.x);

    float ms; // elapsed time in milliseconds
    // create events and streams
    hipEvent_t startEvent, stopEvent, dummyEvent;
    hipStream_t stream[nStreams];
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventCreate(&dummyEvent);
    for (int i = 0; i < nStreams; ++i)
        hipStreamCreate(&stream[i]);

    hipEventRecord(startEvent,0);
    hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
    kernel<<<grid, block>>>(d_a, value);
    hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("Time for sequential transfer and execute (ms): %f\n", ms);

    bool bFinalResults = (bool) checkResult(h_a, num, value);
    printf("result:%s\n", bFinalResults? "PASS":"FAILED");
    // end
    free(h_a);
    hipFree(d_a);
     // cleanup
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipEventDestroy(dummyEvent);
    for (int i = 0; i < nStreams; ++i)
        hipStreamDestroy(stream[i]);

    hipEvent_t events[64];
    for (int i = 0; i < 64; ++i){
        // create
        hipEventCreate(&events[i]);
    }
    for (int i = 0; i < 64; ++i){
        hipEventDestroy(events[i]);
    }
    
    printf("long long int size = %d\n", sizeof(long long int));
    printf("size of hipEvent_t = %d\n", sizeof(hipEvent_t));
    return 0;
}
